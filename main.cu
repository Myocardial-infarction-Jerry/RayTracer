#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <stdio.h>

#include "utils.h"
#include "material.h"
#include "sphere.h"
#include "camera.h"
#include "hittable_list.h"
#include "bvh.h"
#include "texture.h"

__global__ void randInit(hiprandState *randState) {
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;
    hiprand_init(RAND_SEED, 0, 0, randState);
}

__global__ void renderInit(int maxX, int maxY, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= maxX || j >= maxY)
        return;

    int pixelIdx = j * maxX + i;
    hiprand_init(RAND_SEED + pixelIdx, 0, 0, &randState[pixelIdx]);
}

__device__ vec3 getColor(const ray &r, hittable **world, hiprandState *localRandState) {
    ray curRay = r;
    vec3 curAttenuation = vec3(1.0f, 1.0f, 1.0f);
    vec3 color = vec3(0.0f, 0.0f, 0.0f);
    for (int depth = 0; depth < RAY_DEPTH; ++depth) {
        hitRecord rec;
        if (!(*world)->hit(curRay, interval(0.001f, FLT_MAX), rec)) {
            // vec3 unitDirection = curRay.direction().unit();
            // float t = 0.5f * (unitDirection.y() + 1.0f);
            // vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
            // color += curAttenuation * c * 1;
            color += curAttenuation * ((hittable_list *)(*world))->background;
            return color;
        }

        ray scattered;
        vec3 attenuation = vec3(0.0f, 0.0f, 0.0f);
        vec3 colorFromEmission = rec.matPtr->emitted(0.0, 0.0, attenuation);
        color += colorFromEmission * curAttenuation;
        if (!rec.matPtr->scatter(curRay, rec, attenuation, scattered, localRandState))
            return color;

        curAttenuation *= attenuation;
        curRay = scattered;
    }

    return vec3(0.0f, 0.0f, 0.0f);
}

__global__ void render(vec3 *fb, int maxX, int maxY, int ns, camera **cam, hittable **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= maxX || j >= maxY)
        return;

    int pixelIdx = j * maxX + i;
    hiprandState localRandState = randState[pixelIdx];
    vec3 color(0, 0, 0);
    for (int s = 0; s < ns; ++s) {
        ray r = (*cam)->getRay(float(i + RND) / float(maxX), float(j + RND) / float(maxY), &localRandState);
        color += getColor(r, world, &localRandState);
    }

    randState[pixelIdx] = localRandState;
    color /= float(ns);
    color[0] = sqrt(color[0]);
    color[1] = sqrt(color[1]);
    color[2] = sqrt(color[2]);
    fb[pixelIdx] = color;
}

__global__ void freeWorld(hittable **dWorld, camera **dCamera) {
    for (hittable *cur = (*dWorld)->nextObject; cur != nullptr; cur = cur->nextObject)
        delete cur;
    delete *dWorld;
    delete *dCamera;
}

__global__ void randomSphere(hittable **dWorld, camera **dCamera, int nx, int ny, hiprandState *randState) {
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;

    hiprandState localRandState = *randState;
    *dWorld = new hittable_list();
    ((hittable_list *)(*dWorld))->background = vec3(0.5f, 0.7f, 1.0f) * 0.2f;
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(0.0f, -10000.0f, -1.0f), 10000.0f, new lambertian(new checkerTexture(0.32f, vec3(.2f, .3f, .1f), vec3(.9f, .9f, .9f)))));

    for (int i = -11; i < 11; ++i)
        for (int j = -11; j < 11; ++j) {
            float chooseMat = RND;
            vec3 center(i + RND, 0.2, j + RND);

            if (chooseMat < 0.8f)
                ((hittable_list *)(*dWorld))->add(new sphere(center, center + vec3(0, RND * 0.5, 0), 0.2, new lambertian(vec3(RND * RND, RND * RND, RND * RND))));
            else if (chooseMat < 0.95f)
                ((hittable_list *)(*dWorld))->add(new sphere(center, 0.2, new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND)));
            else
                ((hittable_list *)(*dWorld))->add(new sphere(center, 0.2, new dielectric(1.5)));
        }
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5)));
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1))));
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0)));
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(4, 8, 3), 3, new diffuseLight(vec3(1, .9, .6) * 10.0f)));
    *randState = localRandState;
    // (*dWorld) = (hittable_list *)(new bvhNode(*dWorld));

    vec3 lookFrom(0, 2, 14);
    vec3 lookAt(0, 0, 0);
    float focusLen = 10.0f;
    float aperture = 0.01f;
    *dCamera = new camera(
        lookFrom,
        lookAt,
        vec3(0, 1, 0),
        30.0,
        float(nx) / float(ny),
        aperture,
        focusLen
    );
}

__global__ void twoSphere(hittable **dWorld,camera **dCamera,int nx,int ny,hiprandState *randState){
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;

    hiprandState localRandState = *randState;
    *dWorld = new hittable_list();
    ((hittable_list *)(*dWorld))->background = vec3(0.5f, 0.7f, 1.0f);

    auto checker = new checkerTexture(0.8f, vec3(.2f, .3f, .1f), vec3(.9f, .9f, .9f));
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(0, -10, 0), 10, new lambertian(checker)));
    ((hittable_list *)(*dWorld))->add(new sphere(vec3(0, 10, 0), 10, new lambertian(checker)));
    *randState = localRandState;

    vec3 lookFrom(13, 2, 3);
    vec3 lookAt(0, 0, 0);
    float focusLen = 10.0f;
    float aperture = 0.1f;
    *dCamera = new camera(
        lookFrom,
        lookAt,
        vec3(0, 1, 0),
        30.0,
        float(nx) / float(ny),
        aperture,
        focusLen
    );
}

int main(int argc, char const *argv[]) {
    int nx = IMAGE_WIDTH;
    int ny = IMAGE_HEIGHT;
    int ns = SAMPLE_PER_PIXEL;
    int tx = 16;
    int ty = 16;
    size_t stackSize = 2048;

    checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize, stackSize));
    std::cerr << "CUDA Stack Size Limit: " << stackSize << " bytes\n";

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int pixelNum = nx * ny;
    size_t fbSize = pixelNum * sizeof(vec3);

    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fbSize));

    hiprandState *dRandState;
    checkCudaErrors(hipMalloc((void **)&dRandState, pixelNum * sizeof(hiprandState)));
    hiprandState *dRandState_;
    checkCudaErrors(hipMalloc((void **)&dRandState_, 1 * sizeof(hiprandState)));

    randInit << <1, 1 >> > (dRandState_);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hittable **dWorld;
    checkCudaErrors(hipMalloc((void **)&dWorld, sizeof(hittable *)));
    camera **dCamera;
    checkCudaErrors(hipMalloc((void **)&dCamera, sizeof(camera *)));
    switch (0) {
    case 0:
        randomSphere << <1, 1 >> > (dWorld, dCamera, nx, ny, dRandState_);
        break;
    case 1:
        twoSphere << <1, 1 >> > (dWorld, dCamera, nx, ny, dRandState_);
        break;
    }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto start = std::chrono::high_resolution_clock::now();

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    renderInit << <blocks, threads >> > (nx, ny, dRandState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, nx, ny, ns, dCamera, dWorld, dRandState);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    std::cerr << "\ntook " << (float)(std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()) / 1000.0f << " s.\n";

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixelIdx = j * nx + i;
            int ir = (int)(fmin(256, 255.99 * fb[pixelIdx].r()));
            int ig = (int)(fmin(256, 255.99 * fb[pixelIdx].g()));
            int ib = (int)(fmin(256, 255.99 * fb[pixelIdx].b()));
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    freeWorld << <1, 1 >> > (dWorld, dCamera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(dCamera));
    checkCudaErrors(hipFree(dWorld));
    checkCudaErrors(hipFree(dRandState));
    checkCudaErrors(hipFree(dRandState_));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();

    return 0;
}
